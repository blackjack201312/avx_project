#include "hip/hip_runtime.h"
/*
 * nn.cu
 * Nearest Neighbor
 *
 */

#include <stdio.h>
#include <sys/time.h>
#include <float.h>
#include <vector>
#include "hip/hip_runtime.h"

#define min( a, b )			a > b ? b : a
#define ceilDiv( a, b )		( a + b - 1 ) / b
#define print( x )			printf( #x ": %lu\n", (unsigned long) x )
#define DEBUG				false

#define DEFAULT_THREADS_PER_BLOCK 256

#define MAX_ARGS 10
#define REC_LENGTH 53 // size of a record in db
#define REC_WINDOW 1024*128 // number of records to read at a time
#define LATITUDE_POS 28	// character position of the latitude value in each record
#define OPEN 10000	// initial value of nearest neighbors


typedef struct latLong
{
  float lat;
  float lng;
} LatLong;

typedef struct record
{
  char recString[REC_LENGTH];
  float distance;
} Record;

int loadData(char *filename,std::vector<Record> &records,std::vector<LatLong> &locations);
void findLowest(std::vector<Record> &records,float *distances,int numRecords,int topN);
void printUsage();
int parseCommandline(int argc, char *argv[], char* filename,int *r,float *lat,float *lng,
                     int *q, int *t, int *p, int *d);

/**
* Kernel
* Executed on GPU
* Calculates the Euclidean distance from each record in the database to the target position
*/
__global__ void euclid(LatLong *d_locations, float *d_distances, int numRecords,float lat, float lng)
{
	//int globalId = gridDim.x * blockDim.x * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
	int globalId = blockDim.x * ( gridDim.x * blockIdx.y + blockIdx.x ) + threadIdx.x; // more efficient
    LatLong *latLong = d_locations+globalId;
    if (globalId < numRecords) {
        float *dist=d_distances+globalId;
        *dist = (float)sqrt((lat-latLong->lat)*(lat-latLong->lat)+(lng-latLong->lng)*(lng-latLong->lng));
	}
}

/**
* This program finds the k-nearest neighbors
**/

int main(int argc, char* argv[])
{
	int    i=0;
	float lat, lng;
	int quiet=0,timing=0,num_iter_control=1,dev_id=0;

    std::vector<Record> records;
	std::vector<LatLong> locations;
	char filename[100];
	int resultsCount=10;

    // parse command line
    if (parseCommandline(argc, argv, filename,&resultsCount,&lat,&lng,
                     &quiet, &timing, &num_iter_control, &dev_id)) {
      printUsage();
      return 0;
    }
 
 hipDeviceProp_t deviceProp;
  printf("Device ID is %d, Loop is %d \n",dev_id,num_iter_control);
  printf("Choosing CUDA Device....\n");
  hipError_t set_result = hipSetDevice(dev_id);
  printf("Set Result is: %s\n",hipGetErrorString(set_result));
  hipGetDevice(&dev_id);
  hipGetDeviceProperties(&deviceProp, dev_id);
  printf("Name:                     %s\n", deviceProp.name);   

    int numRecords = loadData(filename,records,locations);
    if (resultsCount > numRecords) resultsCount = numRecords;

    printf("Deal with %d records.\n", numRecords);

    //for(i=0;i<numRecords;i++)
    //  printf("%s, %f, %f\n",(records[i].recString),locations[i].lat,locations[i].lng);


    //Pointers to host memory
	float *distances;
	//Pointers to device memory
	LatLong *d_locations;
	float *d_distances;


	// Scaling calculations - added by Sam Kauffman
	hipDeviceSynchronize();
	unsigned long maxGridX = deviceProp.maxGridSize[0];
	unsigned long threadsPerBlock = min( deviceProp.maxThreadsPerBlock, DEFAULT_THREADS_PER_BLOCK );
	size_t totalDeviceMemory;
	size_t freeDeviceMemory;
	hipMemGetInfo(  &freeDeviceMemory, &totalDeviceMemory );
	hipDeviceSynchronize();
	unsigned long usableDeviceMemory = freeDeviceMemory * 85 / 100; // 85% arbitrary throttle to compensate for known CUDA bug
	unsigned long maxThreads = usableDeviceMemory / 12; // 4 bytes in 3 vectors per thread
	if ( numRecords > maxThreads )
	{
		fprintf( stderr, "Error: Input too large.\n" );
		exit( 1 );
	}
	unsigned long blocks = ceilDiv( numRecords, threadsPerBlock ); // extra threads will do nothing
	unsigned long gridY = ceilDiv( blocks, maxGridX );
	unsigned long gridX = ceilDiv( blocks, gridY );
	// There will be no more than (gridY - 1) extra blocks
	dim3 gridDim( gridX, gridY );

	if ( DEBUG )
	{
		print( totalDeviceMemory ); // 804454400
		print( freeDeviceMemory );
		print( usableDeviceMemory );
		print( maxGridX ); // 65535
		print( deviceProp.maxThreadsPerBlock ); // 1024
		print( threadsPerBlock );
		print( maxThreads );
		print( blocks ); // 130933
		print( gridY );
		print( gridX );
	}

	/**
	* Allocate memory on host and device
	*/
	distances = (float *)malloc(sizeof(float) * numRecords);
	hipMalloc((void **) &d_locations,sizeof(LatLong) * numRecords);
	hipMalloc((void **) &d_distances,sizeof(float) * numRecords);

   /**
    * Transfer data from host to device
    */
    hipMemcpy( d_locations, &locations[0], sizeof(LatLong) * numRecords, hipMemcpyHostToDevice);

    /**
    * Execute kernel
    */
float elapsedTime;
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);

for(int i=0;i<num_iter_control;i++){
    euclid<<< gridDim, threadsPerBlock >>>(d_locations,d_distances,numRecords,lat,lng);
    hipDeviceSynchronize();
}
hipEventRecord(stop,0);
hipDeviceSynchronize();
hipEventElapsedTime(&elapsedTime, start, stop);
printf("Kernal Elapsed time : %lf ms\n",elapsedTime);
hipEventDestroy(start);
hipEventDestroy(stop);


    //Copy data from device memory to host memory
    hipMemcpy( distances, d_distances, sizeof(float)*numRecords, hipMemcpyDeviceToHost );

	// find the resultsCount least distances
    findLowest(records,distances,numRecords,resultsCount);

    // print out results
    if (!quiet)
    for(i=0;i<resultsCount;i++) {
      printf("%s --> Distance=%f\n",records[i].recString,records[i].distance);
    }
    free(distances);
    //Free memory
	hipFree(d_locations);
	hipFree(d_distances);

}

int loadData(char *filename,std::vector<Record> &records,std::vector<LatLong> &locations){
    FILE   *flist,*fp;
	int    i=0;
	char dbname[564];
	int recNum=0;

    /**Main processing **/
    flist = fopen(filename, "r");
	while(!feof(flist)) {
		/**
		* Read in all records of length REC_LENGTH
		* If this is the last file in the filelist, then done
		* else open next file to be read next iteration
		*/
		if(fscanf(flist, "%s\n", dbname) != 1) {
            fprintf(stderr, "error reading filelist\n");
            exit(0);
        }
        fp = fopen(dbname, "r");
        if(!fp) {
            printf("error opening a db\n");
            exit(1);
        }
        // read each record
        while(!feof(fp)){
            Record record;
            LatLong latLong;
            fgets(record.recString,49,fp);
            fgetc(fp); // newline
            if (feof(fp)) break;

            // parse for lat and long
            char substr[6];

            for(i=0;i<5;i++) substr[i] = *(record.recString+i+28);
            substr[5] = '\0';
            latLong.lat = atof(substr);

            for(i=0;i<5;i++) substr[i] = *(record.recString+i+33);
            substr[5] = '\0';
            latLong.lng = atof(substr);

            locations.push_back(latLong);
            records.push_back(record);
            recNum++;
	    if (recNum == REC_WINDOW)
		break;
        }
        fclose(fp);
    }
    fclose(flist);
//    for(i=0;i<rec_count*REC_LENGTH;i++) printf("%c",sandbox[i]);
    return recNum;
}

void findLowest(std::vector<Record> &records,float *distances,int numRecords,int topN){
  int i,j;
  float val;
  int minLoc;
  Record *tempRec;
  float tempDist;

  for(i=0;i<topN;i++) {
    minLoc = i;
    for(j=i;j<numRecords;j++) {
      val = distances[j];
      if (val < distances[minLoc]) minLoc = j;
    }
    // swap locations and distances
    tempRec = &records[i];
    records[i] = records[minLoc];
    records[minLoc] = *tempRec;

    tempDist = distances[i];
    distances[i] = distances[minLoc];
    distances[minLoc] = tempDist;

    // add distance to the min we just found
    records[i].distance = distances[i];
  }
}

int parseCommandline(int argc, char *argv[], char* filename,int *r,float *lat,float *lng,
                     int *q, int *t, int *iter, int *d){
    int i;
    if (argc < 2) return 1; // error
    strncpy(filename,argv[1],100);
    char flag;

    for(i=1;i<argc;i++) {
      if (argv[i][0]=='-') {// flag
        flag = argv[i][1];
          switch (flag) {
            case 'r': // number of results
              i++;
              *r = atoi(argv[i]);
              break;
            case 'l': // lat or lng
              if (argv[i][2]=='a') {//lat
                *lat = atof(argv[i+1]);
              }
              else {//lng
                *lng = atof(argv[i+1]);
              }
              i++;
              break;
            case 'h': // help
              return 1;
            case 'q': // quiet
              *q = 1;
              break;
            case 't': // timing
              *t = 1;
              break;
            case 'i': // platform
	      i++;
              *iter = atoi(argv[i]);
	      printf("Setting Iteration as: %d \n",*iter);
              break;
            case 'd': // device
              i++;
	      *d = atoi(argv[i]);
              printf("Setting dev_id as: %d \n",*d);
              break;
        }
      }
    }
      return 0;
}

void printUsage(){
  printf("Nearest Neighbor Usage\n");
  printf("\n");
  printf("nearestNeighbor [filename] -r [int] -lat [float] -lng [float] [-hqt] [-i [int] -d [int]]\n");
  printf("\n");
  printf("example:\n");
  printf("$ ./nearestNeighbor filelist.txt -r 5 -lat 30 -lng 90\n");
  printf("\n");
  printf("filename     the filename that lists the data input files\n");
  printf("-r [int]     the number of records to return (default: 10)\n");
  printf("-lat [float] the latitude for nearest neighbors (default: 0)\n");
  printf("-lng [float] the longitude for nearest neighbors (default: 0)\n");
  printf("\n");
  printf("-h, --help   Display the help file\n");
  printf("-q           Quiet mode. Suppress all text output.\n");
  printf("-t           Print timing information.\n");
  printf("\n");
  printf("-i [int]     Choose the iteration time\n");
  printf("-d [int]     Choose the device \n");
  printf("\n");
  printf("\n");
  printf("Notes: 1. The filename is required as the first parameter.\n");
}
